
#include <hip/hip_runtime.h>
// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -triple amdgcn -fcuda-is-device -std=c++11 -fvisibility hidden -fapply-global-visibility-to-externs \
// RUN:     -emit-llvm -o - -x hip %s | FileCheck -check-prefixes=HIPDEV %s
// RUN: %clang_cc1 -triple x86_64 -std=c++11 \
// RUN:     -emit-llvm -o - -x hip %s | FileCheck -check-prefixes=HIPHOST %s
// RUN: %clang_cc1 -triple amdgcn -fcuda-is-device -std=c++11 -fvisibility hidden -fapply-global-visibility-to-externs \
// RUN:     -O3 -emit-llvm -o - -x hip %s | FileCheck -check-prefixes=HIPDEVUNSED %s

struct textureReference {
  int a;
};

template <class T, int texType, int hipTextureReadMode>
struct texture : public textureReference {
texture() { a = 1; }
};

__attribute__((hip_pinned_shadow)) texture<float, 2, 1> tex;
// CUDADEV-NOT: @tex
// CUDAHOST-NOT: call i32 @__hipRegisterVar{{.*}}@tex
// HIPDEV: @tex = external addrspace(1) global %struct.texture
// HIPDEV-NOT: declare{{.*}}void @_ZN7textureIfLi2ELi1EEC1Ev
// HIPHOST:  define{{.*}}@_ZN7textureIfLi2ELi1EEC1Ev
// HIPHOST:  call i32 @__hipRegisterVar{{.*}}@tex{{.*}}i32 0, i32 4, i32 0, i32 0)
// HIPDEVUNSED: @tex = external addrspace(1) global %struct.texture
// HIPDEVUNSED-NOT: declare{{.*}}void @_ZN7textureIfLi2ELi1EEC1Ev
